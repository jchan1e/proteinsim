

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cstring>

#include <iostream>
#include <fstream>
#include <vector>

//#include "CSCIx229.h"
//#include <SDL.h>
//#include <SDL_opengl.h>
//#include "objects.h"

using namespace std;

////////////////////

int compMode = 0; // 0 = single thread
                  // 1 = multi thread
                  // 2 = GPU

///////////////////////////////////

__global__ void physics(const int n, const int frame, const float* aminosPrev, float* aminosNext, float* history)
{
  float k = 1.0; // Bond Spring Constant
  float ke = -0.01; // Electrostatic Constant
  float kh = -0.2; // Hydrophobicity Constant
  float kc = 1.0;  // Collision Force Constant

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    float hx = 0.0;
    float hy = 0.0;
    float hz = 0.0;
    float ex = 0.0;
    float ey = 0.0;
    float ez = 0.0;
    float cx = 0.0;
    float cy = 0.0;
    float cz = 0.0;
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    for (int j=0; j < n; ++j)
    {
      if (i != j)
      {
        //calculate forces
        float x1 = aminosPrev[8*i + 0];
        float y1 = aminosPrev[8*i + 1];
        float z1 = aminosPrev[8*i + 2];
        float x2 = aminosPrev[8*j + 0];
        float y2 = aminosPrev[8*j + 1];
        float z2 = aminosPrev[8*j + 2];
        float dx = x2-x1;
        float dy = y2-y1;
        float dz = z2-z1;
        float dist = sqrt(dx*dx + dy*dy + dz*dz);
        float h1 = aminosPrev[8*i + 6];
        float e1 = aminosPrev[8*i + 7];
        float h2 = aminosPrev[8*j + 6];
        float e2 = aminosPrev[8*j + 7];

        float vx = dx/dist;
        float vy = dy/dist;
        float vz = dz/dist;

        // Hydrophobic forces
        // Fh = Kh*h1*h2/(r^14-r^8)
        float d = max(dist, 1.0f);
        hx = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * vx;
        hy = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * vy;
        hz = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * vz;

        // Electrsostatic forces
        // Fe = k*q1*q2/r^2
        ex = ke*e1*e2/min(dist*dist, 1.0f) * vx;
        ey = ke*e1*e2/min(dist*dist, 1.0f) * vy;
        ez = ke*e1*e2/min(dist*dist, 1.0f) * vz;

        // Collision forces
        // soft collisions, spring force model
        cx = 0.0;
        cy = 0.0;
        cz = 0.0;
        if (dist < 1.0)
        {
          cx = kc*(1.0-dist) * vx;
          cy = kc*(1.0-dist) * vy;
          cz = kc*(1.0-dist) * vz;
        }

        //if (hx*hx + hy*hy + hz*hz > 0.01)
        //  cout << "H" << i << ":\t"<< hx << "\t" << hy << "\t" << hz << "\t" << dist << endl;
        //if (ex*ex + ey*ey + ez*ez > 0.01)
        //  cout << "E" << i << ":\t"<< ex << "\t" << ey << "\t" << ez << "\t" << dist << endl;
        //if (cx*cx + cy*cy + cz*cz > 0.01)
        //  cout << "C" << i << ":\t"<< cx << "\t" << cy << "\t" << cz << "\t" << dist << endl;

        fx += hx + ex - cx;
        fy += hy + ey - cy;
        fz += hz + ez - cz;
      }
    }
    // spring tension
    if (i > 0)
    {
      float x1 = aminosPrev[8*i + 0];
      float y1 = aminosPrev[8*i + 1];
      float z1 = aminosPrev[8*i + 2];
      float x2 = aminosPrev[8*(i-1) + 0];
      float y2 = aminosPrev[8*(i-1) + 1];
      float z2 = aminosPrev[8*(i-1) + 2];
      float dx = x2-x1;
      float dy = y2-y1;
      float dz = z2-z1;
      float dist = sqrt(dx*dx + dy*dy + dz*dz);
      fx += k*(dist-1.0) * dx/dist;
      fy += k*(dist-1.0) * dy/dist;
      fz += k*(dist-1.0) * dz/dist;
    }
    if (i < n-1)
    {
      float x1 = aminosPrev[8*i + 0];
      float y1 = aminosPrev[8*i + 1];
      float z1 = aminosPrev[8*i + 2];
      float x2 = aminosPrev[8*(i+1) + 0];
      float y2 = aminosPrev[8*(i+1) + 1];
      float z2 = aminosPrev[8*(i+1) + 2];
      float dx = x2-x1;
      float dy = y2-y1;
      float dz = z2-z1;
      float dist = sqrt(dx*dx + dy*dy + dz*dz);
      fx += k*(dist-1.0) * dx/dist;
      fy += k*(dist-1.0) * dy/dist;
      fz += k*(dist-1.0) * dz/dist;
    }

    // update velocities
    aminosNext[8*i+3] = aminosPrev[8*i+3] + fx;
    aminosNext[8*i+4] = aminosPrev[8*i+4] + fy;
    aminosNext[8*i+5] = aminosPrev[8*i+5] + fz;
    // damping
    aminosNext[8*i+3] *= 0.9995;
    aminosNext[8*i+4] *= 0.9995;
    aminosNext[8*i+5] *= 0.9995;
    // update positions
    aminosNext[8*i+0] = aminosPrev[8*i+0] + 0.1*aminosNext[8*i+3];
    aminosNext[8*i+1] = aminosPrev[8*i+1] + 0.1*aminosNext[8*i+4];
    aminosNext[8*i+2] = aminosPrev[8*i+2] + 0.1*aminosNext[8*i+5];
    // copy to history
    history[3*n*frame + 3*i + 0] = aminosNext[8*i+0];
    history[3*n*frame + 3*i + 1] = aminosNext[8*i+1];
    history[3*n*frame + 3*i + 2] = aminosNext[8*i+2];
  }
}

///////////////////////////////////

//void dummy(int n, float* nodes, float* hist)
//{
//  for (int i=0; i < n; ++i)
//  {
//    float hx = 0.0;
//    float hy = 0.0;
//    float hz = 0.0;
//    float ex = 0.0;
//    float ey = 0.0;
//    float ez = 0.0;
//    float cx = 0.0;
//    float cy = 0.0;
//    float cz = 0.0;
//    float fx = 0.0;
//    float fy = 0.0;
//    float fz = 0.0;
//    for (int j=0; j < n; ++j)
//    {
//      if (i != j)
//      {
//        float x1 = nodes[8*i + 0];
//        float y1 = nodes[8*i + 1];
//        float z1 = nodes[8*i + 2];
//        float x2 = nodes[8*j + 0];
//        float y2 = nodes[8*j + 1];
//        float z2 = nodes[8*j + 2];
//        float dx = x2-x1;
//        float dy = y2-y1;
//        float dz = z2-z1;
//        float dist = sqrt(dx*dx + dy*dy + dz*dz);
//        float h1 = nodes[8*i + 6];
//        float e1 = nodes[8*i + 7];
//        float h2 = nodes[8*j + 6];
//        float e2 = nodes[8*j + 7];
//
//        // Hydrophobic forces
//        // Fh = Kh*h1*h2/(r^14-r^8)
//        float d = max(dist, 1.0f);
//        hx = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * dx/dist;
//        hy = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * dy/dist;
//        hz = kh*h1*h2*(pow(d,-14) - pow(d,-8)) * dz/dist;
//
//        // Electrsostatic forces
//        // Fe = k*q1*q2/r^2
//        ex = ke*e1*e2/min(dist*dist, 1.0f) * dx/dist;
//        ey = ke*e1*e2/min(dist*dist, 1.0f) * dy/dist;
//        ez = ke*e1*e2/min(dist*dist, 1.0f) * dz/dist;
//
//        // Collision forces
//        // soft collisions, spring force model
//        if (dist < 1.0)
//        {
//          cx = kc*(1.0-dist) * dx/dist;
//          cy = kc*(1.0-dist) * dy/dist;
//          cz = kc*(1.0-dist) * dz/dist;
//        }
//        else
//        {
//          cx = 0.0;
//          cy = 0.0;
//          cz = 0.0;
//        }
//
//        //if (hx*hx + hy*hy + hz*hz > 0.01)
//        //  cout << "H" << i << ":\t"<< hx << "\t" << hy << "\t" << hz << "\t" << dist << endl;
//        //if (ex*ex + ey*ey + ez*ez > 0.01)
//        //  cout << "E" << i << ":\t"<< ex << "\t" << ey << "\t" << ez << "\t" << dist << endl;
//        //if (cx*cx + cy*cy + cz*cz > 0.01)
//        //  cout << "C" << i << ":\t"<< cx << "\t" << cy << "\t" << cz << "\t" << dist << endl;
//
//        fx += hx + ex - cx;
//        fy += hy + ey - cy;
//        fz += hz + ez - cz;
//      }
//    }
//    // update velocities
//    nodes[8*i + 3] += fx;
//    nodes[8*i + 4] += fy;
//    nodes[8*i + 5] += fz;
//  }
//  // Spring Tension
//  for (int i=0; i < n-1; ++i)
//  {
//    int j = i + 1;
//    float x1 = nodes[8*i + 0];
//    float y1 = nodes[8*i + 1];
//    float z1 = nodes[8*i + 2];
//    float x2 = nodes[8*j + 0];
//    float y2 = nodes[8*j + 1];
//    float z2 = nodes[8*j + 2];
//    float dx = x2-x1;
//    float dy = y2-y1;
//    float dz = z2-z1;
//    float dist = sqrt(dx*dx + dy*dy + dz*dz);
//    nodes[8*i + 3] += k*(dist-1.0) * dx/dist;
//    nodes[8*i + 4] += k*(dist-1.0) * dy/dist;
//    nodes[8*i + 5] += k*(dist-1.0) * dz/dist;
//    nodes[8*j + 3] -= k*(dist-1.0) * dx/dist;
//    nodes[8*j + 4] -= k*(dist-1.0) * dy/dist;
//    nodes[8*j + 5] -= k*(dist-1.0) * dz/dist;
//
//    //if (dist < 0.9 || dist > 1.25)
//    //  cout << dist << endl;
//  }
//  for (int i=0; i < n; ++i)
//  {
//    // damping
//    nodes[8*i + 3] *= 0.9995;
//    nodes[8*i + 4] *= 0.9995;
//    nodes[8*i + 5] *= 0.9995;
//    // update positions
//    nodes[8*i + 0] += 0.1*nodes[8*i + 3];
//    nodes[8*i + 1] += 0.1*nodes[8*i + 4];
//    nodes[8*i + 2] += 0.1*nodes[8*i + 5];
//    //hist->push_back(nodes[8*i + 0]);
//    //hist->push_back(nodes[8*i + 1]);
//    //hist->push_back(nodes[8*i + 2]);
//  }
//}

int main(int argc, char *argv[])
{
  // flags
  for (int i=0; i < argc; ++i)
  {
    if (strcmp(argv[i],"-m") == 0)
    {
      compMode = 1;
      for (int j=i+1; j < argc; ++j)
      {
        argv[j-1] = argv[j];
      }
      i--;
      argc--;
    }
    if (strcmp(argv[i],"-g") == 0)
    {
      compMode = 2;
      for (int j=i+1; j < argc; ++j)
      {
        argv[j-1] = argv[j];
      }
      i--;
      argc--;
    }
  }

  // args
  if (argc != 3 && argc != 4)
  {
    cerr << "Usage: sim infile outfile [num_frames]\n";
    return 1;
  }

  //Initialize
  int num_frames = 6000;
  if (argc == 4) num_frames = stoi(argv[3]);
  ifstream infile(argv[1]);
  if (!infile.is_open())
  {
    cerr << "could not open file " << argv[1] << endl;
    return 1;
  }
  string line;
  getline(infile, line);
  int nAminos = stoi(line);
  cout << nAminos << endl;
  float* aminos = new float[nAminos*8];
  for (int i=0; i < nAminos; ++i)
  {
    aminos[8*i + 0] = nAminos/2.0 - i; // x coordinate
    aminos[8*i + 1] = 0.0;           // y coordinate
    aminos[8*i + 2] = 0.0;           // z coordinate
    aminos[8*i + 3] = 0.0;           // x velocity
    aminos[8*i + 4] = 0.0;           // y velocity
    aminos[8*i + 5] = 0.0;           // z velocity
    getline(infile, line);
    aminos[8*i + 6] = stof(line);    // hydrophobicity
    getline(infile, line);
    aminos[8*i + 7] = stof(line);    // electrostatic charge
  }
  infile.close();
  aminos[1] = 0.1;
  aminos[5] = 0.01;
  //aminos[nAminos*8-7] = -0.1;
  //aminos[nAminos*8-3] = -0.01;

  //vector<float> history;
  float* history = new float[3*nAminos*num_frames];
  int frames = 0;

  float* g_aminos1 = NULL;
  float* g_aminos2 = NULL;
  float* g_history = NULL;
  hipMalloc(&g_aminos1, 8*nAminos*sizeof(float));
  hipMalloc(&g_aminos2, 8*nAminos*sizeof(float));
  hipMalloc(&g_history, 3*num_frames*nAminos*sizeof(float));

  hipMemcpy(g_aminos1, aminos, 8*nAminos*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(g_aminos2, aminos, 8*nAminos*sizeof(float), hipMemcpyHostToDevice);

  ////////Main Loop////////
  while (frames < num_frames)
  {
    physics<<<max(1,nAminos/128),128>>>(nAminos, frames, g_aminos1, g_aminos2, g_history);
//    float test[8];
//    cudaMemcpy(&test, g_aminos2, 8*sizeof(float), cudaMemcpyDeviceToHost);
//    for (int i=0; i < 8; ++i)
//      cout << test[i] << " ";
//    cout << endl;
    frames += 1;
    float* tmp = g_aminos1;
    g_aminos1 = g_aminos2;
    g_aminos2 = tmp;
  }

  // wait and copy back history
  hipMemcpy(history, g_history, 3*num_frames*nAminos*sizeof(float), hipMemcpyDeviceToHost);

//  for (int i=0; i < frames; ++i)
//  {
//    cout << history[3*nAminos*i + 0 + 0] << " ";
//    cout << history[3*nAminos*i + 0 + 1] << " ";
//    cout << history[3*nAminos*i + 0 + 2] << endl;
//  }

  // write to file
  ofstream outfile(argv[2], ofstream::binary);
  if (!outfile.is_open())
    cerr << "could not open file: " << argv[2] << endl;
  else
  {
    float aminoList[2*nAminos];
    for (int i=0; i < nAminos; ++i)
    {
      aminoList[2*i]   = aminos[8*i + 6];
      aminoList[2*i+1] = aminos[8*i + 7];
    }
    outfile.write((char*)&nAminos, sizeof(int));
    outfile.write((char*)&frames, sizeof(int));
    outfile.write((char*)aminoList, 2*nAminos*sizeof(float));
    outfile.write((char*)history, 3*frames*nAminos*sizeof(float));
    outfile.close();
  }

  //cout << "Shutting Down\n";

  delete[] aminos;
  delete[] history;
  hipFree(g_aminos1);
  hipFree(g_aminos2);
  hipFree(g_history);

  return 0;
}
